#include "hip/hip_runtime.h"
#include <torch/extension.h>


__global__ void d_mult(float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i] * (*b);
    }
}


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

const int block_size = 128;


__forceinline__ int calc_grid_size(int m) {
    return (m + block_size - 1) / block_size;
}


torch::Tensor mult(torch::Tensor a, torch::Tensor b) {
    CHECK_INPUT(a);

    auto c = torch::empty_like(a);
    int n = a.numel();

    d_mult<<<calc_grid_size(n), block_size>>>(
        a.data_ptr<float>(),
        b.data_ptr<float>(),
        c.data_ptr<float>(),
        n
    );

    return c;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("prod", &mult, "Custom vector multiply");
}
